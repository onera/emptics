
#include <hip/hip_runtime.h>
#include <cstdio>

__global__
void kernel_hello_world() {
  printf("Hello World from GPU!\n");
}


int main() {
  kernel_hello_world<<<1,1>>>();
  hipDeviceSynchronize(); // make sure the kernel has finished
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("Error during GPU computation. Error code: %i, Error string: %s\n",error, hipGetErrorString(error));
  }
  return 0;
}
